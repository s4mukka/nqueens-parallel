
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

char **initializeBoard(int length) {
  int i, j;
  char **board = (char **) malloc(length * sizeof(char *));

  for (i = 0; i < length; i++)
    board[i] = (char *) malloc(length * sizeof(char));

  for (i = 0; i < length; i++)
    for (j = 0; j < length; j++)
      board[i][j] = ' ';

  return board;
}

void printBoard(char **board, int length) {
  int i, j;

  printf("Board %dx%d\n", length, length);

  for (i = 0; i < length; i++) {
    for (j = 0; j < length; j++)
      printf("[%c]", board[i][j]);
    printf("\n");
  }
}
