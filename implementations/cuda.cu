// #include "implementation.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TRUE 1
#define FALSE 0

__device__ int isSafe(char **board, int length, int row, int column) {
  int i, j;

  for (i = 0; i < column; i++)
    if (board[row][i] == 'Q')
      return FALSE;

  for (i = row, j = column; i >= 0 && j >= 0; i--, j--)
    if (board[i][j] == 'Q')
        return FALSE;

  for (i = row, j = column; j >= 0 && i < length; i++, j--)
    if (board[i][j] == 'Q')
      return FALSE;

  return TRUE;
}

__device__ void nQueens(char **board, int length, int column, int *solutions) {
  int i;

  if (column >= length) {
    (*solutions)++;
    return;
  }

  for (i = 0; i < length; i++)
    if (isSafe(board, length, i, column)) {
      board[i][column] = 'Q';
      nQueens(board, length, column + 1, solutions);
      board[i][column] = ' ';
    }
}

__global__ void solve(char **board, int length, int column, int *solutions) {
  nQueens(board, length, column, solutions);
}

void solveNQueens(char **board, int length) {
  int solutions = 0;
  int *deviceSolutions;
  char *deviceBoard;

  hipMallocManaged((void **) &deviceSolutions, sizeof(int));
  hipMemcpy(deviceSolutions, &solutions, sizeof(int), hipMemcpyHostToDevice);

  hipMallocManaged((void **) &deviceBoard, length * length * sizeof(char));
  hipMemcpy(deviceBoard, board, length * length * sizeof(char), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(length, length);
  
  solve<<<1, threadsPerBlock>>>(&deviceBoard, length, 0, &solutions);

  printf("Number of solutions: %d\n", solutions);
}
