
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <locale.h>

__device__ int deviceSolutions = 0;

void printTimeDifference(struct timeval t_ini, struct timeval t_fim);
__device__ int isSafe(char *board, int length, int row, int column);
__global__ void nQueens(char *board, int length, int column);
__device__ void nQueensR(char *board, int length, int column);
void solveNQueens(char *board, int length);
char *initializeBoard(int length);
void printBoard(char *board, int length);
__device__ void printBoardD(char *board, int length);

__device__ char * my_strcpy(char *dest, const char *src, int length){
  int i = 0;

  for(i = 0; i < length; i++)
    dest[i] = src[i];

  return dest;
}

int main(int argc, char **argv) {
  struct timeval t_ini, t_fim;
  gettimeofday(&t_ini, NULL);

  int n = strtol(argv[1], NULL, 10);

  char *chessboard = initializeBoard(n);

  // solveNQueens(chessboard, n);

   int solutions = 0;
  char *boardDevice;

  hipMalloc((void **) &boardDevice, n * n * sizeof(char));
  hipMemcpy(boardDevice, chessboard, n * n * sizeof(char), hipMemcpyHostToDevice);

  size_t limit = limit = 1024 * n * n;

  hipDeviceSetLimit(hipLimitStackSize, limit);

  nQueens<<<1, n>>>(boardDevice, n, 0);

  hipMemcpyFromSymbol(&solutions, HIP_SYMBOL(deviceSolutions), sizeof(int));

  setlocale(LC_NUMERIC, "");

  hipDeviceSynchronize();

  printf("Number of solutions: %'d\n", solutions);

  gettimeofday(&t_fim, NULL);
  printTimeDifference(t_ini, t_fim);

  hipFree(boardDevice);

  return 0;
}

void printTimeDifference(struct timeval t_ini, struct timeval t_fim) {
  double d_time = t_fim.tv_sec + t_fim.tv_usec / 1e6
                    - t_ini.tv_sec - t_ini.tv_usec / 1e6;

  printf("Time: %.2fs\n", d_time);
}

#define TRUE 1
#define FALSE 0

__device__ void nQueensR(char *board, int length, int column) {
  int i;

  if (column >= length) {
    atomicAdd(&deviceSolutions, 1);
    return;
  }

  for (i = 0; i < length; i++) {
    if (isSafe(board, length, i, column)) {
      board[i * length + column] = 'Q';
      nQueensR(board, length, column + 1);
      board[i * length + column] = ' ';
    }
  }
}

// __device__ void nQueensR(char *board, int length, int column) {
//   int i, *lastRow;

//   lastRow = (int *) malloc(length * sizeof(int));

//   for(i = 0; i < length; i++)
//     lastRow[i] = 0;

//   lastRow[column]++;

//   printBoardD(board, length);

//   while (1) {
    
//     if (column < 0) {
//       return;
//     }

//     if (column >= length) {
//       // printBoardD(board, length);
//       column--;
//       board[i * length + column] = ' ';
//       if (lastRow[column] < length - 1)
//         lastRow[column]++;
//       else {
//         lastRow[column] = 0;
//         column--;
//         if (column >= 0) {
//           board[lastRow[column] * length + column] = ' ';
//           lastRow[column]++;
//         }
//       }
//       atomicAdd(&deviceSolutions, 1);;
//     }

//     for (i = lastRow[column]; i < length; i++) {
//       if (isSafe(board, length, i, column)) {
//         board[i * length + column] = 'Q';
//         lastRow[column++] = i;
//         break;
//       } else if (i == (length - 1)) {
//         lastRow[column] = 0;
//         column--;

//         if (column >= 0) {
//           board[lastRow[column] * length + column] = ' ';
//           if (lastRow[column] < length - 1)
//             lastRow[column]++;
//           else {
//             lastRow[column] = 0;
//             column--;
//             if (column >= 0) {
//               board[lastRow[column] * length + column] = ' ';
//               lastRow[column]++;
//             }
//           }
//         }
//       }
//     }
//   }
// }

__global__ void nQueens(char *board, int length, int column) {
  int i =  blockIdx.x * blockDim.x + threadIdx.x;

  char *copyBoard = (char *) malloc(length * length * sizeof(char));
  copyBoard = my_strcpy(copyBoard, board, length * length);

  copyBoard[i * length + column] = 'Q';
  // copyBoard[i] = 'Q';
  nQueensR(copyBoard, length, column + 1);
  copyBoard[i * length + column] = ' ';

  free(copyBoard);
}

// void solveNQueens(char *board, int length) {
//   int solutions = 0;
//   char **boardDevice;

//   cudaMalloc(boardDevice, length * length * sizeof(char));
//   cudaMemcpy(boardDevice, board, length * length * sizeof(char), cudaMemcpyHostToDevice);

//   nQueens<<<1, length>>>(boardDevice, length, 0);



//   setlocale(LC_NUMERIC, "");

//   printf("Number of solutions: %'d\n", solutions);
// }

__device__ int isSafe(char *board, int length, int row, int column) {
  int i, j;

  for (i = 0; i < column; i++)
    if (board[row * length + i] == 'Q')
      return FALSE;

  for (i = row, j = column; i >= 0 && j >= 0; i--, j--)
    if (board[i * length + j] == 'Q')
        return FALSE;

  for (i = row, j = column; j >= 0 && i < length; i++, j--)
    if (board[i * length + j] == 'Q')
      return FALSE;

  return TRUE;
}

char *initializeBoard(int length) {
  int i, j;
  char *board = (char *) malloc(length * length * sizeof(char));

  for (i = 0; i < length; i++)
    for (j = 0; j < length; j++)
      board[i * length + j] = ' ';

  return board;
}

void printBoard(char *board, int length) {
  int i, j;

  printf("Board %dx%d\n", length, length);

  for (i = 0; i < length; i++) {
    for (j = 0; j < length; j++)
      printf("[%c]", board[i * length + j]);
    printf("\n");
  }
}

__device__ void printBoardD(char *board, int length) {
  int i, j;

  printf("Board %dx%d #%d (sum=%d)\n", length, length, blockIdx.x * blockDim.x + threadIdx.x, deviceSolutions);

  for (i = 0; i < length; i++) {
    for (j = 0; j < length; j++)
      printf("[%c]", board[i * length + j]);
    printf("\n");
  }
}
